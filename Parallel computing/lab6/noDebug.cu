
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

#define M_PI           3.14159265358979323846


//print section
void print_arr(double *arr, int n){
    for(int i = 0; i < n; i++)
    {
        printf("%f ", arr[i]);
    }
    printf("\n");
}

__global__ void randomKernel(unsigned int seed, double *array, int n, int A, int option){
    int tid = blockIdx.x;//blockIdx.x; // * blockDim.x + threadIdx.x;
    hiprandState_t state;

    hiprand_init(seed, tid, 0, &state);
    // Generation for the first Array
    if(option == 1)
    {
        for (int i = tid; i < n; i += blockDim.x * gridDim.x)
        {
            unsigned int randomValue = hiprand(&state);
            array[i] = (randomValue % (A * 100)) / 100.0 + 1;
        }
    }
        // Generation for the second Array
    else
    {
        for (int i = tid; i < n; i += blockDim.x * gridDim.x)
        {
            unsigned int randomValue = hiprand(&state);
            array[i] = (A + randomValue % (A * 9));
        }
    }
}

__global__ void copyKernel(double *arr2, double *arr2Copy, int M2){
    int tid =  blockIdx.x;//blockIdx.x; // * blockDim.x + threadIdx.x;
    if(tid < M2)
    {
        arr2Copy[tid] = arr2[tid];
    }
}

__global__ void mapSqrtCthKernel(double *arr1, int M1){
    int tid =  blockIdx.x;//blockIdx.x; //* blockDim.x + threadIdx.x;
    if(tid < M1)
    {
        arr1[tid] = sqrt(cosh(arr1[tid] / sinh(arr1[tid])));
    }
}

__global__ void mapPiCbrtKernel(double *arr2, double *arr2Copy, int M2){
    int tid =  blockIdx.x; //blockIdx.x; // * blockDim.x + threadIdx.x;
    if(tid == 0)
    {
        arr2[tid] = cbrt(arr2[tid]*M_PI);
    }
    if(tid > 0 &&  tid < M2)
    {
        arr2[tid] = cbrt((arr2[tid] + arr2Copy[tid]) * M_PI);
    }
}

__global__ void multiplayKenrel(double *arr1, double *arr2, int M2){
    int tid =  blockIdx.x; //blockIdx.x; // * blockDim.x + threadIdx.x;
    if(tid < M2)
    {
        arr2[tid] = arr1[tid] * arr2[tid];
    }
}


__global__ void reduceKernel(double *arr2, int M2, float *blockSums){

    int tid = blockIdx.x;//threadIdx.x; // + blockIdx.x * blockDim.x;

    if(tid < M2){
        if(((int)arr2[tid] / (int)arr2[0]) % 2 == 0)
        {
            atomicAdd(blockSums, (float) sin(arr2[tid]));
        }
    }
}


int main(){
    hipError_t error = hipGetLastError();
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    int k = 0;

    int M1, M2;
    int A;
    //M1 = atoi(argv[1]);
    M1 = 806569;
    M2 = M1 / 2;
    A = 100;

    double *arr1;
    double *arr2;
    double *arr2Copy;

    hipMalloc((void**)&arr1, M1 * sizeof (double ));
    hipMalloc((void**)&arr2, M2 * sizeof(double ));
    hipMalloc((void**)&arr2Copy, M2 * sizeof(double));
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }

    int threadsPerBlock = 256;
    int blocksPerGridArr1 = (M1 + threadsPerBlock - 1) / threadsPerBlock;
    int blocksPerGridArr2 = (M2 + threadsPerBlock - 1) / threadsPerBlock;

    hipEventRecord(start);
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
    for(k = 0; k< 100; k++){
        unsigned int seed = k;

        // Creating arrays
        randomKernel<<<blocksPerGridArr1, threadsPerBlock>>>(seed, arr1, M1, A, 1);
        randomKernel<<<blocksPerGridArr2, threadsPerBlock>>>(seed, arr2, M2, A, 2);
        copyKernel<<<blocksPerGridArr2, threadsPerBlock>>>(arr2, arr2Copy, M2);

        // Stage 2 - Map creating
        mapSqrtCthKernel<<<blocksPerGridArr2, threadsPerBlock>>>(arr1, M1);
        mapPiCbrtKernel<<<blocksPerGridArr2, threadsPerBlock>>>(arr2, arr2Copy, M2);

        // Stage  3 - Merge multiply
        multiplayKenrel<<<blocksPerGridArr2, threadsPerBlock>>>(arr1, arr2, M2);

        // Stage 5 - Reduce
        float sum = 0.0f;
        float *deviceSum;
        hipMalloc((void**)&deviceSum, sizeof(float));

        hipMemcpy(deviceSum, &sum, sizeof(float), hipMemcpyHostToDevice);
        reduceKernel<<<blocksPerGridArr2, threadsPerBlock>>>(arr2, M2, deviceSum);
        hipDeviceSynchronize();

        float hostResult;
        hipMemcpy(&hostResult, deviceSum, sizeof(float), hipMemcpyDeviceToHost);

        hipFree(deviceSum);
        printf("%f ", hostResult);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);
    printf("\nExecution time of GPU: %f ms\n", milliseconds);

    hipFree(arr1);
    hipFree(arr2);
    hipFree(arr2Copy);

    return 0;
}