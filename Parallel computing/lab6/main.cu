
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

#define M_PI           3.14159265358979323846
//#define time_event
//#define DEBUG

//print section
void print_arr(double *arr, int n){
    for(int i = 0; i < n; i++)
    {
        printf("%f ", arr[i]);
    }
    printf("\n");
}
//blockIdx.x * blockDim.x + threadIdx.x
__global__ void randomKernel1(unsigned int seed, double *array, int n, int A){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState_t state;
    __shared__ double sharedArray[512];

    hiprand_init(seed, tid, 0, &state);

    if(tid < n){
        sharedArray[threadIdx.x] = array[tid];
    }
    __syncthreads();

    // Generation for the first Array

    if(tid < n){
        unsigned int randomValue = hiprand(&state);
        sharedArray[threadIdx.x] = (randomValue % (A * 100)) / 100.0 + 1;
        //array[i] = (randomValue % (A * 100)) / 100.0 + 1;

    }
    __syncthreads();

    if (tid < n) {
        array[tid] = sharedArray[threadIdx.x];
    }
}
__global__ void randomKernel2(unsigned int seed, double *array, int n, int A){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState_t state;
    __shared__ double sharedArray[512];

    hiprand_init(seed, tid, 0, &state);

    if(tid < n){
        sharedArray[threadIdx.x] = array[tid];
    }
    __syncthreads();

    if(tid < n){
        unsigned int randomValue = hiprand(&state);
        sharedArray[threadIdx.x] = A + randomValue % (A * 9);
        //array[i] = (A + randomValue % (A * 9));
    }

    __syncthreads();

    if (tid < n) {
        array[tid] = sharedArray[threadIdx.x];
    }
}
__global__ void copyKernel(double *arr2, double *arr2Copy, int M2){
    int tid =  blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < M2)
    {
        arr2Copy[tid] = arr2[tid];
    }
}

__global__ void mapSqrtCthKernel(double *arr1, int M1){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < M1)
    {
        arr1[tid] = sqrt(cosh(arr1[tid] / sinh(arr1[tid])));
    }
    __syncthreads();
}

__global__ void mapPiCbrtKernel(double *arr2, double *arr2Copy, int M2){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid == 0)
    {
        arr2[tid] = cbrt(arr2[tid]*M_PI);
    }
    if(tid > 0 &&  tid < M2)
    {
        arr2[tid] = cbrt((arr2[tid] + arr2Copy[tid]) * M_PI);
    }
}

__global__ void multiplayKenrel(double *arr1, double *arr2, int M2){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < M2)
    {
        arr2[tid] = arr1[tid] * arr2[tid];
    }
}


__global__ void reduceKernel(double *arr2, int M2, float *blockSums){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid < M2){
        if(((int)arr2[tid] / (int)arr2[0]) % 2 == 0)
        {
            atomicAdd(blockSums, (float) sin(arr2[tid]));
        }
    }
}


int main(){
#ifdef DEBUG
    printf("> Creating cuda events\n");
#endif
    // Declaration of time events
    hipError_t error = hipGetLastError();
    hipEvent_t start, end;
#ifdef time_event
    cudaEvent_t startGeneration, endGeneration;
    cudaEvent_t startStage2, endStage2;
    cudaEvent_t startStage3, endStage3;
    cudaEvent_t startStage5, endStage5;
#endif
    // Creating time events
    hipEventCreate(&start);
    hipEventCreate(&end);
#ifdef time_event
    cudaEventCreate(&startGeneration);
    cudaEventCreate(&endGeneration);
    cudaEventCreate(&startStage2);
    cudaEventCreate(&endStage2);
    cudaEventCreate(&startStage3);
    cudaEventCreate(&endStage3);
    cudaEventCreate(&startStage5);
    cudaEventCreate(&endStage5);
#endif
#ifdef DEBUG
    printf("- Events are created\n");
#endif

    float milliseconds;
    int k = 0;

    int M1, M2;
    int A;
    //M1 = atoi(argv[1]);
    M1 = 742107;
    M2 = M1 / 2;
    A = 100;

    double *arr1;
    double *arr2;
    double *arr2Copy;
    // Allocate memory on device
    hipMalloc((void**)&arr1, M1 * sizeof (double ));
    hipMalloc((void**)&arr2, M2 * sizeof(double ));
    hipMalloc((void**)&arr2Copy, M2 * sizeof(double));
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }

#ifdef DEBUG
    double *arr1_host = (double*)malloc(M1 * sizeof(double));
    double *arr2_host = (double*)malloc(M2 * sizeof(double));
    double *arr2Copy_host = (double*)malloc(M2 * sizeof(double ));
#endif
#ifdef DEBUG
    printf("- Memory are allocated\n");
#endif

    // Calculate of
    int threadsPerBlock = 512;
    int blocksPerGridArr1 = (M1 + threadsPerBlock - 1) / threadsPerBlock;
    int blocksPerGridArr2 = (M2 + threadsPerBlock - 1) / threadsPerBlock;

    hipEventRecord(start);
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
    for(k = 0; k< 100; k++){
        unsigned int seed = k;

        // Creating arrays
#ifdef time_event
        cudaEventRecord(startGeneration);
#endif
#ifdef DEBUG
        printf("> Initialization of threads block for generator\n");
#endif
#ifdef DEBUG
        printf("> Start of generator\n");
#endif
        randomKernel1<<<blocksPerGridArr1, threadsPerBlock>>>(seed, arr1, M1, A);
        randomKernel2<<<blocksPerGridArr2, threadsPerBlock>>>(seed, arr2, M2, A);
        copyKernel<<<blocksPerGridArr2, threadsPerBlock>>>(arr2, arr2Copy, M2);
#ifdef DEBUG
        printf("- Generation is succeed\n");
                printf("> Closing cores\n");
#endif
#ifdef  DEBUG
        cudaMemcpy(arr1_host, arr1, M1 * sizeof(double), cudaMemcpyDeviceToHost);
        cudaMemcpy(arr2_host, arr2, M2 * sizeof(double), cudaMemcpyDeviceToHost);
        cudaMemcpy(arr2Copy_host, arr2Copy, M2 * sizeof(double), cudaMemcpyDeviceToHost);
        print_arr(arr1_host, M1);
        print_arr(arr2_host, M2);
        print_arr(arr2Copy_host, M2);
#endif
#ifdef time_event
        cudaEventRecord(endGeneration);
        cudaEventSynchronize(endGeneration);
        cudaEventElapsedTime(&milliseconds, startGeneration, endGeneration);
        printf("\nTime of stage 1: %f ms\n", milliseconds);
#endif
        // Stage 2 - Map creating
#ifdef time_event
        cudaEventRecord(startStage2);
#endif
#ifdef DEBUG
        printf("> Stage 2 - Map creating  \n");
#endif
        mapSqrtCthKernel<<<blocksPerGridArr2, threadsPerBlock>>>(arr1, M1);
        mapPiCbrtKernel<<<blocksPerGridArr2, threadsPerBlock>>>(arr2, arr2Copy, M2);
#ifdef  time_event
        cudaEventRecord(endStage2);
        cudaEventSynchronize(endStage2);
        cudaEventElapsedTime(&milliseconds, startStage2, endStage2);
        printf("\nTime of stage 2: %f ms\n", milliseconds);
#endif
#ifdef  DEBUG
        cudaMemcpy(arr1_host, arr1, M1 * sizeof(double), cudaMemcpyDeviceToHost);
        cudaMemcpy(arr2_host, arr2, M2 * sizeof(double), cudaMemcpyDeviceToHost);
        print_arr(arr1_host, M1);
        print_arr(arr2_host, M2);
#endif
#ifdef DEBUG
        printf("> arr2[0] = cbrt(arr2[0]*M_PI); \n");
#endif
        // Stage  3 - Merge multiply
#ifdef time_event
        cudaEventRecord(startStage3);
#endif
#ifdef DEBUG
        printf("> Stage 3 - Merge \n");
#endif
        multiplayKenrel<<<blocksPerGridArr2, threadsPerBlock>>>(arr1, arr2, M2);
#ifdef time_event
        cudaEventRecord(endStage3);
        cudaEventSynchronize(endStage3);
        cudaEventElapsedTime(&milliseconds, startStage3, endStage3);
        printf("\nTime of stage 3: %f ms\n", milliseconds);
#endif
#ifdef DEBUG
        printf("> Stage 3 - Merge is done ! \n");
#endif
#ifdef  DEBUG
        cudaMemcpy(arr2_host, arr2, M2 * sizeof(double), cudaMemcpyDeviceToHost);
            print_arr(arr2_host, M2);
#endif

        // Stage 5 - Reduce
#ifdef time_event
        cudaEventRecord(startStage5);
#endif
#ifdef DEBUG
        printf("> Stage 5 - Reduce \n");
#endif
        float sum = 0.0f;
        float *deviceSum;
        hipMalloc((void**)&deviceSum, sizeof(float));

        hipMemcpy(deviceSum, &sum, sizeof(float), hipMemcpyHostToDevice);
        reduceKernel<<<blocksPerGridArr2, threadsPerBlock>>>(arr2, M2, deviceSum);
        hipDeviceSynchronize();
#ifdef time_event
        cudaEventRecord(endStage5);
        cudaEventSynchronize(endStage5);
        cudaEventElapsedTime(&milliseconds, startStage5, endStage5);
        printf("\nTime of stage 5: %f ms\n", milliseconds);
#endif
#ifdef DEBUG
        printf("- Stage 5 - Reduce is done\n");
#endif

        float hostResult;
        hipMemcpy(&hostResult, deviceSum, sizeof(float), hipMemcpyDeviceToHost);
        hipFree(deviceSum);
        printf("%f ", hostResult);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);
    printf("\nExecution time of GPU: %f ms\n", milliseconds);

    hipFree(arr1);
    hipFree(arr2);
    hipFree(arr2Copy);

    return 0;
}
