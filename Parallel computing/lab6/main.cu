#include <math.h>
#include <stdio.h>
#include "stdlib.h"
#include "time.h"
#include "string.h"
#include "hip/hip_runtime.h"

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
//#include "cuda_runtime_api.h"

#define M_PI           3.14159265358979323846
//#define DEBUG_MERGE
//#define DEBUG_REDUCE
//#define DEBUG



//print section
void print_arr(double *arr, int n){
    for(int i = 0; i < n; i++)
    {
        printf("%f ", arr[i]);
    }
    printf("\n");
}

//swap section
void swap(double *x, double *y) {
    double tmp;
    tmp = *x;
    *x = *y;
    *y = tmp;
}

//merge section
void merge(double *arr, int n){
    double *arr1 = (double*)malloc(n/2 * sizeof(double ));
    double *arr2 =(double *)malloc((n - n/2)* sizeof(double ));
                    //new double [n - n/2];

    for(int i = 0; i < n/2; i++){
        arr1[i] = arr[i];
    }

    #ifdef DEBUG_MERGE
        printf("Merge of sorted arr1 \n");
        print_arr(arr1, n/2);
    #endif

    int current = 0;
    for(int i = n/2; i < n; i++){
        arr2[current] = arr[i];
        current += 1;
    }

    #ifdef DEBUG_MERGE
    printf("Merge of sorted arr2 \n");
        print_arr(arr2, n - n/2);
    #endif

    int n1 = n/2;
    int n2= n - n/2;
    int l = 0;
    int r = 0;
    current = 0;

    while(l < n1 && r < n2){
        if(arr1[l] <= arr2[r]){
            arr[current] = arr1[l];
            l += 1;
        } else {
            arr[current] = arr2[r];
            r += 1;
        }
        current += 1;
    }

    while(l < n1){
        arr[current] = arr1[l];
        l += 1;
        current += 1;
    }

    while(r < n2) {
        arr[current] = arr2[r];
        r += 1;
        current += 1;
    }
    free(arr1);
    free(arr2);
}


__global__ void randomKernel(unsigned int seed, double *array, int n, int A, int option){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState_t state;

    hiprand_init(seed, tid, 0, &state);
    // Generation for the first Array
    if(option == 1)
    {
        for (int i = tid; i < n; i += blockDim.x * gridDim.x)
        {
            unsigned int randomValue = hiprand(&state);
            array[i] = (randomValue % (A * 100)) / 100.0 + 1;
        }
    }
    // Generation for the second Array
    else
    {
        for (int i = tid; i < n; i += blockDim.x * gridDim.x)
        {
            unsigned int randomValue = hiprand(&state);
            array[i] = (A + randomValue % (A * 9));
        }
    }

}

__global__ void copyKernel(double *arr2, double *arr2Copy, int M2){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < M2)
    {
        arr2Copy[tid] = arr2[tid];
    }
}

__global__ void mapSqrtCthKernel(double *arr1, int M1){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < M1)
    {
        arr1[tid] = sqrt(cosh(arr1[tid] / sinh(arr1[tid])));
    }
}

__global__ void mapPiCbrtKernel(double *arr2, double *arr2Copy, int M2){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid == 0)
    {
        arr2[tid] = cbrt(arr2[tid]*M_PI);
    }
    if(tid > 0 &&  tid < M2)
    {
        arr2[tid] = cbrt((arr2[tid] + arr2Copy[tid]) * M_PI);
    }
}

__global__ void multiplayKenrel(double *arr1, double *arr2, int M2){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < M2)
    {
        arr2[tid] = arr1[tid] * arr2[tid];
    }
}

__global__ void stupidSortKernel(double *array, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int half = n / 2;

    double *leftArray = array;
    double *rightArray = array + half;

    if (tid < half) {
        int i = 0;
        while (i < half - 1) {
            if (leftArray[i + 1] < leftArray[i]) {
                double temp = leftArray[i];
                leftArray[i] = leftArray[i + 1];
                leftArray[i + 1] = temp;
                i = 0;
            } else {
                i++;
            }
        }
    } else {
        int i = 0;
        while (i < n - half - 1) {
            if (rightArray[i + 1] < rightArray[i]) {
                double temp = rightArray[i];
                rightArray[i] = rightArray[i + 1];
                rightArray[i + 1] = temp;
                i = 0;
            } else {
                i++;
            }
        }
    }
    __syncthreads();

    if (tid < n - half) {
        array[tid] = leftArray[tid];
    } else {
        array[tid] = rightArray[tid - half];
    }
}
__constant__ double minValue;

__global__ void reduceKernel(double *arr2, int M2, float *blockSums){
    __shared__ float sum;
    sum = 0;

#ifdef DEBUG_REDUCE
    printf("Start sum is: %f\n", sum);
#endif

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
#ifdef DEBUG_REDUCE
    printf("Tid is: %d\n", tid);
    printf("Stride is: %d\n", stride);
#endif

    if(tid < M2){
        if(((int)arr2[tid] / (int)arr2[0]) % 2 == 0)
        {
            atomicAdd(blockSums, (float)arr2[tid]);
        }
    }
}

//int argc, char *argv[]
int main() {
    #ifdef DEBUG
            printf("> Creating cuda events\n");
    #endif

    hipError_t error = hipGetLastError();
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    #ifdef DEBUG
        printf("- Events are created\n");
    #endif

    int k = 0;

    int M1, M2;
    int A;
    //M1 = atoi(argv[1]);
    M1 = 1000;
    M2 = M1 / 2;
    A = 100;


    double *arr1;
    double *arr2;
    double *arr2Copy;

    #ifdef DEBUG
        printf("> Allocate memory for arrays\n");
    #endif

    hipMalloc((void**)&arr1, M1 * sizeof (double ));
    hipMalloc((void**)&arr2, M2 * sizeof(double ));
    hipMalloc((void**)&arr2Copy, M2 * sizeof(double));
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
    double *arr1_host = (double*)malloc(M1 * sizeof(double));
    double *arr2_host = (double*)malloc(M2 * sizeof(double));
    double *arr2Copy_host = (double*)malloc(M2 * sizeof(double ));;

    #ifdef DEBUG
        double *arr1_host = (double*)malloc(M1 * sizeof(double));
        double *arr2_host = (double*)malloc(M2 * sizeof(double));
        double *arr2Copy_host = (double*)malloc(M2 * sizeof(double ));;
    #endif

    #ifdef DEBUG
        printf("- Memory are allocated\n");
    #endif


    hipEventRecord(start);
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
    for(k = 0; k < 100; k++) {
        unsigned int seed = k;

        // Arrays generation
        #ifdef DEBUG
                printf("> Initialization of threads block for generator\n");
        #endif

        int threadsPerBlock = 1024;
        int blocksPerGridArr1 = (M1 + threadsPerBlock - 1) / threadsPerBlock;
        int blocksPerGridArr2 = (M2 + threadsPerBlock - 1) / threadsPerBlock;
        #ifdef DEBUG
                printf("> Start of generator\n");
        #endif

        // Creating arrays
        randomKernel<<<blocksPerGridArr1, threadsPerBlock>>>(seed, arr1, M1, A, 1);
        randomKernel<<<blocksPerGridArr2, threadsPerBlock>>>(seed, arr2, M2, A, 2);
        copyKernel<<<blocksPerGridArr2, threadsPerBlock>>>(arr2, arr2Copy, M2);

        #ifdef DEBUG
                printf("- Generation is succeed\n");
                printf("> Closing cores\n");
        #endif
        hipDeviceSynchronize();

        #ifdef  DEBUG
            cudaMemcpy(arr1_host, arr1, M1 * sizeof(double), cudaMemcpyDeviceToHost);
            cudaMemcpy(arr2_host, arr2, M2 * sizeof(double), cudaMemcpyDeviceToHost);
            print_arr(arr1_host, M1);
            print_arr(arr2_host, M2);
        #endif

        // Stage 2 - Map creating
        #ifdef DEBUG
            printf("> Stage 2 - Map creating  \n");
        #endif
        mapSqrtCthKernel<<<blocksPerGridArr2, threadsPerBlock>>>(arr1, M1);
        mapPiCbrtKernel<<<blocksPerGridArr2, threadsPerBlock>>>(arr2, arr2Copy, M2);
        #ifdef DEBUG
                printf("- Stage 2 - Map is done !\n");
        #endif
        hipDeviceSynchronize();
        #ifdef DEBUG
                printf("> arr2[0] = cbrt(arr2[0]*M_PI); \n");
        #endif
        //arr2[0] = cbrt(arr2[0]*M_PI);

        #ifdef  DEBUG
            cudaMemcpy(arr1_host, arr1, M1 * sizeof(double), cudaMemcpyDeviceToHost);
            cudaMemcpy(arr2_host, arr2, M2 * sizeof(double), cudaMemcpyDeviceToHost);
            print_arr(arr1_host, M1);
            print_arr(arr2_host, M2);
        #endif
        // Stage  3 - Merge multiply
        #ifdef DEBUG
                printf("> Stage 3 - Merge \n");
        #endif
        multiplayKenrel<<<blocksPerGridArr2, threadsPerBlock>>>(arr1, arr2, M2);
        #ifdef DEBUG
                printf("> Stage 3 - Merge is done ! \n");
        #endif
        hipDeviceSynchronize();

        #ifdef  DEBUG
            cudaMemcpy(arr2_host, arr2, M2 * sizeof(double), cudaMemcpyDeviceToHost);
            print_arr(arr2_host, M2);
        #endif

        // Stage 4 - sort
        #ifdef DEBUG
                printf("> Stage 4 - Sort \n");
        #endif
        stupidSortKernel<<<blocksPerGridArr2, threadsPerBlock>>>(arr2, M2);
        hipDeviceSynchronize();
        #ifdef DEBUG
                printf("- Stage 4 - Sort is done\n");
        #endif
        #ifdef  DEBUG
                cudaMemcpy(arr2_host, arr2, M2 * sizeof(double), cudaMemcpyDeviceToHost);
                print_arr(arr2_host, M2);
        #endif
        hipMemcpy(arr2_host, arr2, M2 * sizeof(double), hipMemcpyDeviceToHost);
        #ifdef DEBUG
                printf("> Merge of sorted array\n");
        #endif
        merge(arr2_host, M2);
        #ifdef DEBUG
                printf("- Merge of sorted array is done\n");
                print_arr(arr2_host, M2);
        #endif
        hipMemcpy(arr2, arr2_host, M2 * sizeof(double), hipMemcpyHostToDevice);
        #ifdef DEBUG
                printf("- From host to device arr2copy\n");
        #endif
        // Stage 5 - Reduce
        float sum = 0.0f;
        float *deviceSum;
        hipMalloc((void**)&deviceSum, sizeof(float));

        #ifdef DEBUG
                printf("> Stage 5 - Reduce \n");
        #endif

        hipMemcpy(deviceSum, &sum, sizeof(float), hipMemcpyHostToDevice);
        reduceKernel<<<blocksPerGridArr2, threadsPerBlock>>>(arr2, M2, deviceSum);
        hipDeviceSynchronize();

        float hostResult;
        hipMemcpy(&hostResult, deviceSum, sizeof(float), hipMemcpyDeviceToHost);

        #ifdef DEBUG
                printf("- Stage 5 - Reduce is done\n");
        #endif

        hipFree(deviceSum);
        printf("%f ", hostResult);

    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    hipFree(arr1);
    hipFree(arr2);
    hipFree(arr2Copy);
    free(arr1_host);
    free(arr2_host);
    free(arr2Copy_host);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);
    printf("\nExecution time of GPU: %f ms\n", milliseconds);
    return 0;
}
